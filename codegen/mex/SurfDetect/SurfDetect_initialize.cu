/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * SurfDetect_initialize.cu
 *
 * Code generation for function 'SurfDetect_initialize'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "SurfDetect.h"
#include "SurfDetect_initialize.h"
#include "_coder_SurfDetect_mex.h"
#include "SurfDetect_data.h"

/* Function Definitions */
void SurfDetect_initialize(void)
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

/* End of code generation (SurfDetect_initialize.cu) */
