/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * SurfDetect_terminate.cu
 *
 * Code generation for function 'SurfDetect_terminate'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "SurfDetect.h"
#include "SurfDetect_terminate.h"
#include "_coder_SurfDetect_mex.h"
#include "SurfDetect_data.h"

/* Function Definitions */
void SurfDetect_atexit(void)
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

void SurfDetect_terminate(void)
{
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (SurfDetect_terminate.cu) */
