/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_SurfDetect_api.cu
 *
 * Code generation for function '_coder_SurfDetect_api'
 *
 */

/* Include files */
#include "rt_nonfinite.h"
#include "SurfDetect.h"
#include "_coder_SurfDetect_api.h"
#include "SurfDetect_data.h"

/* Function Declarations */
static uint8_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[589824];
static uint8_T (*c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId))[589824];
static uint8_T (*emlrt_marshallIn(const mxArray *inputImage, const char_T
  *identifier))[589824];
static const mxArray *emlrt_marshallOut(const struct0_T u_data[], const int32_T
  u_size[2]);

/* Function Definitions */
static uint8_T (*b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[589824]
{
  uint8_T (*y)[589824];
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}
  static uint8_T (*c_emlrt_marshallIn(const mxArray *src, const
  emlrtMsgIdentifier *msgId))[589824]
{
  uint8_T (*ret)[589824];
  static const int32_T dims[3] = { 384, 512, 3 };

  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "uint8", false, 3U,
    dims);
  ret = (uint8_T (*)[589824])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static uint8_T (*emlrt_marshallIn(const mxArray *inputImage, const char_T
  *identifier))[589824]
{
  uint8_T (*y)[589824];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = (const char *)identifier;
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(inputImage), &thisId);
  emlrtDestroyArray(&inputImage);
  return y;
}
  static const mxArray *emlrt_marshallOut(const struct0_T u_data[], const
  int32_T u_size[2])
{
  const mxArray *y;
  int32_T iv1[2];
  int32_T i;
  static const char * sv0[5] = { "x", "y", "scale", "orientation", "laplacian" };

  int32_T b_j1;
  const mxArray *b_y;
  const mxArray *m0;
  y = NULL;
  for (i = 0; i < 2; i++) {
    iv1[i] = u_size[i];
  }

  emlrtAssign(&y, emlrtCreateStructArray(2, iv1, 5, sv0));
  emlrtCreateField(y, "x");
  emlrtCreateField(y, "y");
  emlrtCreateField(y, "scale");
  emlrtCreateField(y, "orientation");
  emlrtCreateField(y, "laplacian");
  i = 0;
  for (b_j1 = 0; b_j1 < 2000; b_j1++) {
    if (b_j1 < u_size[1U]) {
      b_y = NULL;
      m0 = emlrtCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);
      *(real32_T *)emlrtMxGetData(m0) = u_data[u_size[0] * b_j1].x;
      emlrtAssign(&b_y, m0);
      emlrtSetFieldR2017b(y, i, "x", b_y, 0);
      b_y = NULL;
      m0 = emlrtCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);
      *(real32_T *)emlrtMxGetData(m0) = u_data[u_size[0] * b_j1].y;
      emlrtAssign(&b_y, m0);
      emlrtSetFieldR2017b(y, i, "y", b_y, 1);
      b_y = NULL;
      m0 = emlrtCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);
      *(real32_T *)emlrtMxGetData(m0) = u_data[u_size[0] * b_j1].scale;
      emlrtAssign(&b_y, m0);
      emlrtSetFieldR2017b(y, i, "scale", b_y, 2);
      b_y = NULL;
      m0 = emlrtCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);
      *(real32_T *)emlrtMxGetData(m0) = u_data[u_size[0] * b_j1].orientation;
      emlrtAssign(&b_y, m0);
      emlrtSetFieldR2017b(y, i, "orientation", b_y, 3);
      b_y = NULL;
      m0 = emlrtCreateNumericMatrix(1, 1, mxINT32_CLASS, mxREAL);
      *(int32_T *)emlrtMxGetData(m0) = u_data[u_size[0] * b_j1].laplacian;
      emlrtAssign(&b_y, m0);
      emlrtSetFieldR2017b(y, i, "laplacian", b_y, 4);
      i++;
    }
  }

  return y;
}

void SurfDetect_api(SurfDetectStackData *SD, const mxArray * const prhs[1],
                    const mxArray *plhs[1])
{
  uint8_T (*inputImage)[589824];
  struct0_T intPoints_data[2000];
  int32_T intPoints_size[2];

  /* Marshall function inputs */
  inputImage = emlrt_marshallIn(emlrtAlias(prhs[0]), "inputImage");

  /* Invoke the target function */
  SurfDetect(SD, *inputImage, intPoints_data, intPoints_size);

  /* Marshall function outputs */
  plhs[0] = emlrt_marshallOut(intPoints_data, intPoints_size);
}

/* End of code generation (_coder_SurfDetect_api.cu) */
