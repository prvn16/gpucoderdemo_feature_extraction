/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * _coder_SurfDetect_mex.cu
 *
 * Code generation for function '_coder_SurfDetect_mex'
 *
 */

/* Include files */
#include "SurfDetect.h"
#include "_coder_SurfDetect_mex.h"
#include "SurfDetect_terminate.h"
#include "_coder_SurfDetect_api.h"
#include "SurfDetect_initialize.h"
#include "SurfDetect_data.h"

/* Function Declarations */
static void SurfDetect_mexFunction(SurfDetectStackData *SD, int32_T nlhs,
  mxArray *plhs[1], int32_T nrhs, const mxArray *prhs[1]);

/* Function Definitions */
static void SurfDetect_mexFunction(SurfDetectStackData *SD, int32_T nlhs,
  mxArray *plhs[1], int32_T nrhs, const mxArray *prhs[1])
{
  int32_T n;
  const mxArray *inputs[1];
  const mxArray *outputs[1];
  int32_T b_nlhs;

  /* Check for proper number of arguments. */
  if (nrhs != 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 1, 4, 10, "SurfDetect");
  }

  if (nlhs > 1) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 10,
                        "SurfDetect");
  }

  /* Temporary copy for mex inputs. */
  for (n = 0; n < nrhs; n++) {
    inputs[n] = prhs[n];
  }

  /* Call the function. */
  SurfDetect_api(SD, inputs, outputs);

  /* Copy over outputs to the caller. */
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }

  emlrtReturnArrays(b_nlhs, plhs, outputs);

  /* Module termination. */
  SurfDetect_terminate();
}

void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs, const mxArray
                 *prhs[])
{
  SurfDetectStackData *SurfDetectStackDataGlobal = NULL;
  SurfDetectStackDataGlobal = (SurfDetectStackData *)emlrtMxCalloc(1, 1U *
    sizeof(SurfDetectStackData));
  mexAtExit(SurfDetect_atexit);

  /* Initialize the memory manager. */
  /* Module initialization. */
  SurfDetect_initialize();

  /* Dispatch the entry-point. */
  SurfDetect_mexFunction(SurfDetectStackDataGlobal, nlhs, plhs, nrhs, prhs);
  emlrtMxFree(SurfDetectStackDataGlobal);
}

emlrtCTX mexFunctionCreateRootTLS(void)
{
  emlrtCreateRootTLS(&emlrtRootTLSGlobal, &emlrtContextGlobal, NULL, 1);
  return emlrtRootTLSGlobal;
}

/* End of code generation (_coder_SurfDetect_mex.cu) */
