//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: SurfDetect_initialize.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 09:58:13
//

// Include Files
#include "rt_nonfinite.h"
#include "SurfDetect.h"
#include "SurfDetect_initialize.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void SurfDetect_initialize()
{
  rt_InitInfAndNaN(8U);
}

//
// File trailer for SurfDetect_initialize.cu
//
// [EOF]
//
