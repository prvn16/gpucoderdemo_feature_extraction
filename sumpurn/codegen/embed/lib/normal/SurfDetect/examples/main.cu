//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: main.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 23-Jan-2018 09:58:13
//

//***********************************************************************
// This automatically generated example C main file shows how to call
// entry-point functions that MATLAB Coder generated. You must customize
// this file for your application. Do not modify this file directly.
// Instead, make a copy of this file, modify it, and integrate it into
// your development environment.
//
// This file initializes entry-point function arguments to a default
// size and value before calling the entry-point functions. It does
// not store or use any values returned from the entry-point functions.
// If necessary, it does pre-allocate memory for returned values.
// You can use this file as a starting point for a main function that
// you can deploy in your application.
//
// After you copy the file, and before you deploy it, you must make the
// following changes:
// * For variable-size function arguments, change the example sizes to
// the sizes that your application requires.
// * Change the example values of function arguments to the values that
// your application requires.
// * If the entry-point functions return values, store these values or
// otherwise use them as required by your application.
//
//***********************************************************************
// Include Files
#include "rt_nonfinite.h"
#include "SurfDetect.h"
#include "main.h"
#include "SurfDetect_terminate.h"
#include "SurfDetect_initialize.h"

// Function Declarations
static void argInit_384x512x3_uint8_T(uint8_T result[589824]);
static uint8_T argInit_uint8_T();
static void main_SurfDetect();

// Function Definitions

//
// Arguments    : uint8_T result[589824]
// Return Type  : void
//
static void argInit_384x512x3_uint8_T(uint8_T result[589824])
{
  int32_T idx0;
  int32_T idx1;
  int32_T idx2;

  // Loop over the array to initialize each element.
  for (idx0 = 0; idx0 < 384; idx0++) {
    for (idx1 = 0; idx1 < 512; idx1++) {
      for (idx2 = 0; idx2 < 3; idx2++) {
        // Set the value of the array element.
        // Change this value to the value that the application requires.
        result[(idx0 + 384 * idx1) + 196608 * idx2] = argInit_uint8_T();
      }
    }
  }
}

//
// Arguments    : void
// Return Type  : uint8_T
//
static uint8_T argInit_uint8_T()
{
  return 0;
}

//
// Arguments    : void
// Return Type  : void
//
static void main_SurfDetect()
{
  struct0_T intPoints_data[2000];
  int32_T intPoints_size[2];
  static uint8_T b[589824];

  // Initialize function 'SurfDetect' input arguments.
  // Initialize function input argument 'inputImage'.
  // Call the entry-point 'SurfDetect'.
  argInit_384x512x3_uint8_T(b);
  SurfDetect(b, intPoints_data, intPoints_size);
}

//
// Arguments    : int32_T argc
//                const char * const argv[]
// Return Type  : int32_T
//
int32_T main(int32_T, const char * const [])
{
  // Initialize the application.
  // You do not need to do this more than one time.
  SurfDetect_initialize();

  // Invoke the entry-point functions.
  // You can call entry-point functions multiple times.
  main_SurfDetect();

  // Terminate the application.
  // You do not need to do this more than one time.
  SurfDetect_terminate();
  return 0;
}

//
// File trailer for main.cu
//
// [EOF]
//
